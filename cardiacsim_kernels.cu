
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define BLOCK_SIZE 16

double *d_E, *d_R, *d_E_prev;

void mirror_halos(double **mat, int m, int n);

__global__ void v1_PDE(double *E, double *E_prev, double *R,
					   const double alpha, const int n, const int m, const double kk,
					   const double dt, const double a, const double epsilon,
					   const double M1, const double M2, const double b)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int width = m + 2;

	int index = row * width + col;

	if (row <= m && col <= n)
		E[index] = E_prev[index] + alpha * (E_prev[index + 1] + E_prev[index - 1] - 4 * E_prev[index] + E_prev[index + width] + E_prev[index - width]);
}

__global__ void v1_ODE(double *E, double *E_prev, double *R,
					   const double alpha, const int n, const int m, const double kk,
					   const double dt, const double a, const double epsilon,
					   const double M1, const double M2, const double b)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int index = row * (m + 2) + col;

	if (row <= m && col <= n)
	{
		E[index] = E[index] - dt * (kk * E[index] * (E[index] - a) * (E[index] - 1) + E[index] * R[index]);
		R[index] = R[index] + dt * (epsilon + M1 * R[index] / (E[index] + M2)) * (-R[index] - kk * E[index] * (E[index] - b - 1));
	}
}

__global__ void v2_kernel(double *E, double *E_prev, double *R,
						  const double alpha, const int n, const int m, const double kk,
						  const double dt, const double a, const double epsilon,
						  const double M1, const double M2, const double b)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int width = m + 2;

	int index = row * width + col;

	if (row <= m && col <= n)
	{
		E[index] = E_prev[index] + alpha * (E_prev[index + 1] + E_prev[index - 1] - 4 * E_prev[index] + E_prev[index + width] + E_prev[index - width]);

		__syncthreads();

		E[index] = E[index] - dt * (kk * E[index] * (E[index] - a) * (E[index] - 1) + E[index] * R[index]);
		R[index] = R[index] + dt * (epsilon + M1 * R[index] / (E[index] + M2)) * (-R[index] - kk * E[index] * (E[index] - b - 1));
	}
}

__global__ void v3_kernel(double *E, double *E_prev, double *R,
						  const double alpha, const int n, const int m, const double kk,
						  const double dt, const double a, const double epsilon,
						  const double M1, const double M2, const double b)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int width = m + 2;

	int index = row * width + col;

	if (row <= m && col <= n)
	{
		E[index] = E_prev[index] + alpha * (E_prev[index + 1] + E_prev[index - 1] - 4 * E_prev[index] + E_prev[index + width] + E_prev[index - width]);

		__syncthreads();

		double e_current = E[index];
		double r_current = R[index];

		E[index] = e_current - dt * (kk * e_current * (e_current - a) * (e_current - 1) + e_current * r_current);
		e_current = E[index];

		R[index] = r_current + dt * (epsilon + M1 * r_current / (e_current + M2)) * (-r_current - kk * e_current * (e_current - b - 1));
	}
}

void kernel1(double **E, double **E_prev, double **R, const double alpha, const int n, const int m, const double kk,
			 const double dt, const double a, const double epsilon, const double M1, const double M2, const double b, int shouldMalloc, int shouldFree)
{

	mirror_halos(E_prev, m, n);

	int nx = n + 2, ny = m + 2;
	int matSize = sizeof(double) * nx * ny;
	int copyOffset = ny;
	
	if (shouldMalloc)
	{
		hipMalloc(&d_E, matSize);
		hipMalloc(&d_R, matSize);
		hipMalloc(&d_E_prev, matSize);
		hipMemcpy(d_R, &R[0] + copyOffset, matSize, hipMemcpyHostToDevice);
	}

	hipMemcpy(d_E, &E[0] + copyOffset, matSize, hipMemcpyHostToDevice);
	hipMemcpy(d_E_prev, &E_prev[0] + copyOffset, matSize, hipMemcpyHostToDevice);

	const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	int dimension = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	const dim3 grid(dimension, dimension);

	v1_PDE<<<grid, block>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	hipDeviceSynchronize();
	v1_ODE<<<grid, block>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	hipDeviceSynchronize();

	hipMemcpy(E + copyOffset, d_E, matSize, hipMemcpyDeviceToHost);
	hipMemcpy(E_prev + copyOffset, d_E_prev, matSize, hipMemcpyDeviceToHost);
	if (shouldFree)
	{
		hipMemcpy(R + copyOffset, d_R, matSize, hipMemcpyDeviceToHost);
		hipFree(d_E);
		hipFree(d_R);
		hipFree(d_E_prev);
	}
}

void kernel2(double **E, double **E_prev, double **R, const double alpha, const int n, const int m, const double kk,
			 const double dt, const double a, const double epsilon, const double M1, const double M2, const double b, int shouldMalloc, int shouldFree)
{

	mirror_halos(E_prev, m, n);

	int nx = n + 2, ny = m + 2;
	int matSize = sizeof(double) * nx * ny;
	int copyOffset = ny;

	if (shouldMalloc)
	{
		hipMalloc(&d_E, matSize);
		hipMalloc(&d_R, matSize);
		hipMalloc(&d_E_prev, matSize);
		hipMemcpy(d_R, &R[0] + copyOffset, matSize, hipMemcpyHostToDevice);
	}

	hipMemcpy(d_E, &E[0] + copyOffset, matSize, hipMemcpyHostToDevice);
	hipMemcpy(d_E_prev, &E_prev[0] + copyOffset, matSize, hipMemcpyHostToDevice);

	const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	int dimension = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	const dim3 grid(dimension, dimension);

	v2_kernel<<<grid, block>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	hipDeviceSynchronize();

	hipMemcpy(E + copyOffset, d_E, matSize, hipMemcpyDeviceToHost);
	hipMemcpy(E_prev + copyOffset, d_E_prev, matSize, hipMemcpyDeviceToHost);
	if (shouldFree)
	{
		hipMemcpy(R + copyOffset, d_R, matSize, hipMemcpyDeviceToHost);
		hipFree(d_E);
		hipFree(d_R);
		hipFree(d_E_prev);
	}
}

void kernel3(double **E, double **E_prev, double **R, const double alpha, const int n, const int m, const double kk,
			 const double dt, const double a, const double epsilon, const double M1, const double M2, const double b, int shouldMalloc, int shouldFree)
{

	mirror_halos(E_prev, m, n);

	int nx = n + 2, ny = m + 2;
	int matSize = sizeof(double) * nx * ny;
	int copyOffset = ny;

	if (shouldMalloc)
	{
		hipMalloc(&d_E, matSize);
		hipMalloc(&d_R, matSize);
		hipMalloc(&d_E_prev, matSize);
		hipMemcpy(d_R, &R[0] + copyOffset, matSize, hipMemcpyHostToDevice);
	}

	hipMemcpy(d_E, &E[0] + copyOffset, matSize, hipMemcpyHostToDevice);
	hipMemcpy(d_E_prev, &E_prev[0] + copyOffset, matSize, hipMemcpyHostToDevice);

	const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	int dimension = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	const dim3 grid(dimension, dimension);

	v3_kernel<<<grid, block>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	hipDeviceSynchronize();

	hipMemcpy(E + copyOffset, d_E, matSize, hipMemcpyDeviceToHost);
	hipMemcpy(E_prev + copyOffset, d_E_prev, matSize, hipMemcpyDeviceToHost);
	if (shouldFree)
	{
		hipMemcpy(R + copyOffset, d_R, matSize, hipMemcpyDeviceToHost);
		hipFree(d_E);
		hipFree(d_R);
		hipFree(d_E_prev);
	}
}

void mirror_halos(double **mat, int m, int n)
{
	int i, j;

	for (j = 1; j <= m; j++)
	{
		mat[j][0] = mat[j][2];
		mat[j][n + 1] = mat[j][n - 1];
	}

	for (i = 1; i <= n; i++)
	{
		mat[0][i] = mat[2][i];
		mat[m + 1][i] = mat[m - 1][i];
	}
}