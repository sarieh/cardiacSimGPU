
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define BLOCK_SIZE 17

double *d_E, *d_R, *d_E_prev;

void mirror_halos(double **mat, int m, int n);

__global__ void v1_PDE(double *E, double *E_prev, double *R,
					   const double alpha, const int n, const int m, const double kk,
					   const double dt, const double a, const double epsilon,
					   const double M1, const double M2, const double b)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int width = m + 2;

	int index = row * width + col;

	if (row <= m && col <= n)
		E[index] = E_prev[index] + alpha * (E_prev[index + 1] + E_prev[index - 1] - 4 * E_prev[index] + E_prev[index + width] + E_prev[index - width]);
}

__global__ void v1_ODE(double *E, double *E_prev, double *R,
					   const double alpha, const int n, const int m, const double kk,
					   const double dt, const double a, const double epsilon,
					   const double M1, const double M2, const double b)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int index = row * (m + 2) + col;

	if (row <= m && col <= n)
	{
		E[index] = E[index] - dt * (kk * E[index] * (E[index] - a) * (E[index] - 1) + E[index] * R[index]);
		R[index] = R[index] + dt * (epsilon + M1 * R[index] / (E[index] + M2)) * (-R[index] - kk * E[index] * (E[index] - b - 1));
	}
}

__global__ void v2_kernel(double *E, double *E_prev, double *R,
						  const double alpha, const int n, const int m, const double kk,
						  const double dt, const double a, const double epsilon,
						  const double M1, const double M2, const double b)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int width = m + 2;

	int index = row * width + col;

	if (row <= m && col <= n)
	{
		E[index] = E_prev[index] + alpha * (E_prev[index + 1] + E_prev[index - 1] - 4 * E_prev[index] + E_prev[index + width] + E_prev[index - width]);

		__syncthreads();

		E[index] = E[index] - dt * (kk * E[index] * (E[index] - a) * (E[index] - 1) + E[index] * R[index]);
		R[index] = R[index] + dt * (epsilon + M1 * R[index] / (E[index] + M2)) * (-R[index] - kk * E[index] * (E[index] - b - 1));
	}
}

__global__ void v3_kernel(double *E, double *E_prev, double *R,
						  const double alpha, const int n, const int m, const double kk,
						  const double dt, const double a, const double epsilon,
						  const double M1, const double M2, const double b)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int width = m + 2;

	int index = row * width + col;

	if (row <= m && col <= n)
	{

		E[index] = E_prev[index] + alpha * (E_prev[index + 1] + E_prev[index - 1] - 4 * E_prev[index] + E_prev[index + width] + E_prev[index - width]);

		__syncthreads();

		double e_current = E[index];
		double r_current = R[index];

		E[index] = e_current - dt * (kk * e_current * (e_current - a) * (e_current - 1) + e_current * r_current);
		e_current = E[index];

		R[index] = r_current + dt * (epsilon + M1 * r_current / (e_current + M2)) * (-r_current - kk * e_current * (e_current - b - 1));
	}
}

__global__ void v4_kernel(double *E, double *E_prev, double *R,
						  const double alpha, const int n, const int m, const double kk,
						  const double dt, const double a, const double epsilon,
						  const double M1, const double M2, const double b, int bx, int by)
{
	extern __shared__ double shared_E_prev[];

	const int block_width = blockDim.x + 2;

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int l_col = threadIdx.x + 1;
	int l_row = threadIdx.y + 1;

	int gwidth = m + 2;
	int gindex = row * gwidth + col;

	int lindex = l_row * block_width + l_col;

	// Read input elements into shared memory
	shared_E_prev[lindex] = E_prev[gindex];
	//Load ghost cells
	if (threadIdx.x < 1)
	{
		shared_E_prev[lindex - 1] = E_prev[gindex - 1];
		shared_E_prev[lindex + bx] = E_prev[gindex + bx];
	}
	if (threadIdx.y < 1)
	{
		shared_E_prev[lindex - block_width] = E_prev[gindex - gwidth];
		shared_E_prev[lindex + block_width * by] = E_prev[gindex + gwidth * by];
	}

	/*
		x - - - - - - x
		- * x - - x * -
		- * - * * - * -
		- * - * * - * -
		- * x - - x * -
		x - - - - - - x
		*/

	__syncthreads(); // Make sure all threads loaded into the shared memory

	if (row <= m && col <= n)
	{
		//PDE
		double e_current = shared_E_prev[lindex] + alpha * (shared_E_prev[lindex + 1] + shared_E_prev[lindex - 1] - 4 * shared_E_prev[lindex] + shared_E_prev[lindex + block_width] + shared_E_prev[lindex - block_width]);
		double r_current = R[gindex];
		__syncthreads();

		//ODE
		e_current = e_current - dt * (kk * e_current * (e_current - a) * (e_current - 1) + e_current * r_current);
		r_current = r_current + dt * (epsilon + M1 * r_current / (e_current + M2)) * (-r_current - kk * e_current * (e_current - b - 1));

		E[gindex] = e_current;
		R[gindex] = r_current;
	}
}

void deviceKernel(double **E, double **E_prev, double **R, const double alpha, const int n, const int m, const double kk,
				  const double dt, const double a, const double epsilon, const double M1, const double M2, const double b, int shouldMalloc, int shouldFree, int v, int bx, int by)
{

	mirror_halos(E_prev, m, n);

	int nx = n + 2, ny = m + 2;
	int matSize = sizeof(double) * nx * ny;
	int copyOffset = ny;

	if (shouldMalloc)
	{
		hipMalloc(&d_E, matSize);
		hipMalloc(&d_R, matSize);
		hipMalloc(&d_E_prev, matSize);
		hipMemcpy(d_R, &R[0] + copyOffset, matSize, hipMemcpyHostToDevice);
	}

	hipMemcpy(d_E, &E[0] + copyOffset, matSize, hipMemcpyHostToDevice);
	hipMemcpy(d_E_prev, &E_prev[0] + copyOffset, matSize, hipMemcpyHostToDevice);

	const dim3 block(bx, by);
	int dimension_x = (n + bx - 1) / bx;
	int dimension_y = (n + by - 1) / by;
	const dim3 grid(dimension_x, dimension_y);

	const int block_width = bx + 2;
	const int block_height = by + 2;
	const int sharedBlockSize = block_width * block_height * sizeof(double);

	if (v == 1)
	{
		v1_PDE<<<grid, block>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
		hipDeviceSynchronize();
		v1_ODE<<<grid, block>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	}
	else if (v == 2)
	{
		v2_kernel<<<grid, block>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	}
	else if (v == 3)
	{
		v3_kernel<<<grid, block>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	}
	else
	{
		v4_kernel<<<grid, block, sharedBlockSize>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b, bx, by);
	}
	hipDeviceSynchronize();

	hipMemcpy(E + copyOffset, d_E, matSize, hipMemcpyDeviceToHost);
	hipMemcpy(E_prev + copyOffset, d_E_prev, matSize, hipMemcpyDeviceToHost);
	if (shouldFree)
	{
		hipMemcpy(R + copyOffset, d_R, matSize, hipMemcpyDeviceToHost);
		hipFree(d_E);
		hipFree(d_R);
		hipFree(d_E_prev);
	}
}

void mirror_halos(double **mat, int m, int n)
{
	int i, j;

	for (j = 1; j <= m; j++)
	{
		mat[j][0] = mat[j][2];
		mat[j][n + 1] = mat[j][n - 1];
	}

	for (i = 1; i <= n; i++)
	{
		mat[0][i] = mat[2][i];
		mat[m + 1][i] = mat[m - 1][i];
	}
}