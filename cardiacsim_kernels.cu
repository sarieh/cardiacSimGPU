
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define BLOCK_SIZE 16

void mirror_halos(double **mat, int m, int n);

__global__ void v1_PDE(double *E, double *E_prev, double *R,
					   const double alpha, const int n, const int m, const double kk,
					   const double dt, const double a, const double epsilon,
					   const double M1, const double M2, const double b)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int width = m + 2;

	int index = row * width + col;

	if (row <= m && col <= n)
		E[index] = E_prev[index] + alpha * (E_prev[index + 1] + E_prev[index - 1] - 4 * E_prev[index] + E_prev[index + width] + E_prev[index - width]);
}

__global__ void v1_ODE(double *E, double *E_prev, double *R,
					   const double alpha, const int n, const int m, const double kk,
					   const double dt, const double a, const double epsilon,
					   const double M1, const double M2, const double b)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int index = row * (m + 2) + col;

	if (row <= m && col <= n)
	{
		E[index] = E[index] - dt * (kk * E[index] * (E[index] - a) * (E[index] - 1) + E[index] * R[index]);
		R[index] = R[index] + dt * (epsilon + M1 * R[index] / (E[index] + M2)) * (-R[index] - kk * E[index] * (E[index] - b - 1));
	}
}

__global__ void v2_kernel(double *E, double *E_prev, double *R,
						  const double alpha, const int n, const int m, const double kk,
						  const double dt, const double a, const double epsilon,
						  const double M1, const double M2, const double b)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int width = m + 2;

	int index = row * width + col;

	if (row <= m && col <= n)
	{
		E[index] = E_prev[index] + alpha * (E_prev[index + 1] + E_prev[index - 1] - 4 * E_prev[index] + E_prev[index + width] + E_prev[index - width]);

		__syncthreads();

		E[index] = E[index] - dt * (kk * E[index] * (E[index] - a) * (E[index] - 1) + E[index] * R[index]);
		R[index] = R[index] + dt * (epsilon + M1 * R[index] / (E[index] + M2)) * (-R[index] - kk * E[index] * (E[index] - b - 1));
	}
}

__global__ void v3_kernel(double *E, double *E_prev, double *R,
						  const double alpha, const int n, const int m, const double kk,
						  const double dt, const double a, const double epsilon,
						  const double M1, const double M2, const double b)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int width = m + 2;

	int index = row * width + col;

	if (row <= m && col <= n)
	{

		E[index] = E_prev[index] + alpha * (E_prev[index + 1] + E_prev[index - 1] - 4 * E_prev[index] + E_prev[index + width] + E_prev[index - width]);

		__syncthreads();

		double e_current = E[index];
		double r_current = R[index];

		E[index] = e_current - dt * (kk * e_current * (e_current - a) * (e_current - 1) + e_current * r_current);
		e_current = E[index];

		R[index] = r_current + dt * (epsilon + M1 * r_current / (e_current + M2)) * (-r_current - kk * e_current * (e_current - b - 1));
	}
}

__global__ void v4_kernel(double *E, double *E_prev, double *R,
						  const double alpha, const int n, const int m, const double kk,
						  const double dt, const double a, const double epsilon,
						  const double M1, const double M2, const double b)
{
	const int block_width = BLOCK_SIZE + 2;
	const int sharedBlockSize = block_width * block_width;
	__shared__ double shared_E_prev[sharedBlockSize];
	__shared__ double shared_R[sharedBlockSize];

	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int l_col = threadIdx.x + 1;
	int l_row = threadIdx.y + 1;

	int gwidth = m + 2;
	int gindex = row * gwidth + col;

	int lindex = l_row * block_width + l_col;

	// Read input elements into shared memory
	shared_E_prev[lindex] = E_prev[gindex];
	shared_R[lindex] = R[gindex];

	//Load ghost cells
	if (threadIdx.x < 1)
	{
		shared_E_prev[lindex - 1] = E_prev[gindex - 1];
		shared_E_prev[lindex + BLOCK_SIZE] = E_prev[gindex + BLOCK_SIZE];
	}
	if (threadIdx.y < 1)
	{
		shared_E_prev[lindex - block_width] = E_prev[gindex - gwidth];
		shared_E_prev[lindex + block_width * BLOCK_SIZE] = E_prev[gindex + gwidth * BLOCK_SIZE];
	}

	__syncthreads(); // Make sure all threads loaded into the shared memory

	if (row <= m && col <= n)
	{
		//PDE
		double e_current = shared_E_prev[lindex] + alpha * (shared_E_prev[lindex + 1] + shared_E_prev[lindex - 1] - 4 * shared_E_prev[lindex] + shared_E_prev[lindex + block_width] + shared_E_prev[lindex - block_width]);
		double r_current = shared_R[lindex];
		__syncthreads();

		//ODE
		e_current = e_current - dt * (kk * e_current * (e_current - a) * (e_current - 1) + e_current * r_current);
		r_current = r_current + dt * (epsilon + M1 * r_current / (e_current + M2)) * (-r_current - kk * e_current * (e_current - b - 1));

		E[gindex] = e_current;
		R[gindex] = r_current;
	}
}


__global__ void halos_kernel(double *E_prev, const int m, const int n){

	int col = threadIdx.x + 1;
	int row = threadIdx.y + 1;

	int width = m + 2;
	int index = row * width + col;

	if(row == 1){
		E_prev[index - width] = E_prev[index + width];
		E_prev[index + width*m] = E_prev[index + width*(m-2)];
	}
	if(col == 1){
		E_prev[index - 1] = E_prev[index + 1];
		E_prev[index + m] = E_prev[index + (m-2)];
	}
}

void deviceKernel(double **E, double **E_prev, double **R, double **d_E, double **d_E_prev, double **d_R, const double alpha, const int n, const int m, const double kk,
	const double dt, const double a, const double epsilon, const double M1, const double M2, const double b, int shouldMalloc, int shouldFree, int v, int swap)
{

	int nx = n + 2, ny = m + 2;
	int matSize = sizeof(double) * nx * ny;
	int copyOffset = ny;

	if (shouldMalloc)
	{
		hipMalloc(&(*d_E), matSize);
		hipMalloc(&(*d_R), matSize);
		hipMalloc(&(*d_E_prev), matSize);
		hipMemcpy(*d_R, &R[copyOffset], matSize, hipMemcpyHostToDevice);
		hipMemcpy(*d_E, &E[copyOffset], matSize, hipMemcpyHostToDevice);
		hipMemcpy(*d_E_prev, &E_prev[copyOffset], matSize, hipMemcpyHostToDevice);
	}

	const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	int dimension = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	const dim3 grid(dimension, dimension);

	const dim3 halos_block(n, m);
	const dim3 grid_block(1, 1);
	if(swap % 2)
		halos_kernel<<<grid_block, halos_block>>>(*d_E, n, m);
	else
		halos_kernel<<<grid_block, halos_block>>>(*d_E_prev, n, m);
		
	hipDeviceSynchronize();

	if (v == 1)
	{
		if(swap % 2)
			v1_PDE<<<grid, block>>>(*d_E_prev, *d_E, *d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
		else
			v1_PDE<<<grid, block>>>(*d_E, *d_E_prev, *d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);

		hipDeviceSynchronize();
		if(swap % 2)
			v1_ODE<<<grid, block>>>(*d_E_prev, *d_E, *d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
		else
			v1_ODE<<<grid, block>>>(*d_E, *d_E_prev, *d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	}
	else if (v == 2)
	{
		if(swap % 2)
			v2_kernel<<<grid, block>>>(*d_E_prev, *d_E, *d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
		else
			v2_kernel<<<grid, block>>>(*d_E, *d_E_prev, *d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	}
	else if (v == 3)
	{
		if(swap % 2)
			v3_kernel<<<grid, block>>>(*d_E_prev, *d_E, *d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
		else
			v3_kernel<<<grid, block>>>(*d_E, *d_E_prev, *d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	}
	else
	{
		if(swap % 2)
			v4_kernel<<<grid, block>>>(*d_E_prev, *d_E, *d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
		else
			v4_kernel<<<grid, block>>>(*d_E, *d_E_prev, *d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
	}

	hipDeviceSynchronize();
	if (shouldFree)
	{
		if(swap % 2){
			hipMemcpy(&E[copyOffset] , *d_E_prev, matSize, hipMemcpyDeviceToHost);
			hipMemcpy(&E_prev[copyOffset] , *d_E, matSize, hipMemcpyDeviceToHost);
		}else{
			hipMemcpy(&E[copyOffset] , *d_E, matSize, hipMemcpyDeviceToHost);
			hipMemcpy(&E_prev[copyOffset] , *d_E_prev, matSize, hipMemcpyDeviceToHost);	
		}
		
		hipMemcpy(&R[copyOffset] , *d_R, matSize, hipMemcpyDeviceToHost);
		hipFree(*d_E);
		hipFree(*d_R);
		hipFree(*d_E_prev);
	}
}

void mirror_halos(double **mat, int m, int n)
{
	int i, j;

	for (j = 1; j <= m; j++)
	{
		mat[j][0] = mat[j][2];
		mat[j][n + 1] = mat[j][n - 1];
	}

	for (i = 1; i <= n; i++)
	{
		mat[0][i] = mat[2][i];
		mat[m + 1][i] = mat[m - 1][i];
	}
}